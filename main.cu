
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
using namespace std;
using namespace chrono;

// MACROS
#define MIN(x, y) (x < y ? x : y)

string generateWord(int n)
{
	string word = "";
	for (int i = 0; i < n; i++)
	{
		word += (char)('a' + rand() % 26);
	}
	return word;
}

int sequentialDistance(string A, string B)
{
	unsigned int lenA = A.size();
	unsigned int lenB = B.size();

	unsigned int **D = new unsigned int *[lenA + 1];
	for (int i = 0; i < lenA + 1; i++)
		D[i] = new unsigned int[lenB + 1];

	for (int i = 0; i < lenA + 1; i++)
		D[i][0] = i;
	for (int j = 1; j < lenB + 1; j++)
		D[0][j] = j;

	for (int i = 1; i < lenA + 1; i++)
	{
		for (int j = 1; j < lenB + 1; j++)
		{
			if (A[i - 1] == B[j - 1])
			{
				D[i][j] = D[i - 1][j - 1];
			}
			else
			{
				D[i][j] = 1 + min(min(D[i - 1][j], D[i][j - 1]), D[i - 1][j - 1]);
			}
		}
	}

	return D[lenA][lenB];
}

__global__ void editDistKernel(char *devA, char *devB, int lenA, int lenB, unsigned int *devPPrevDiag, unsigned int *devPrevDiag, unsigned int *devCurrDiag, int diagIdx)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int diagSize = (diagIdx <= lenA + 1) ? diagIdx : ((2 * (lenA+1)) - diagIdx);

	/* if (tid == 0) {
		printf("diagSize: %d\n", diagSize);
		for (int j = 0; j <= lenA; j++) printf("%d ", devPPrevDiag[j]);
		printf("-----------\n");
		for (int j = 0; j <= lenA; j++) printf("%d ", devPrevDiag[j]);
		printf("\n");
	} */

	if (tid < diagSize) {
		if (diagIdx <= lenA + 1) {
			if (tid == 0)
				devCurrDiag[tid] = devPrevDiag[tid] + 1;
			if (tid == diagSize - 1)
				devCurrDiag[tid] = devPrevDiag[tid - 1] + 1;

			if (tid > 0 && tid < diagSize - 1) {
				if (devA[tid - 1] != devB[diagSize - tid - 2])
					devCurrDiag[tid] = 1 + MIN(devPrevDiag[tid - 1], MIN(devPPrevDiag[tid - 1], devPrevDiag[tid]));
				else
					devCurrDiag[tid] = devPPrevDiag[tid - 1];
			}
		} else {
			int pprevIdx = (lenA - diagSize == 0) ? tid : tid + 1;
			if (devA[tid + lenA - diagSize] != devB[lenA - tid - 1]) {
				devCurrDiag[tid] = 1 + MIN(devPrevDiag[tid], MIN(devPPrevDiag[pprevIdx], devPrevDiag[tid + 1]));
			}
			else {
				devCurrDiag[tid] = devPPrevDiag[pprevIdx];
			}
		}
	}
}

int parallelDistance(const char *A, const char *B, int lenA, int lenB)
{
	int distance;
	char *devA;
	char *devB;
	unsigned int *devCurrDiag;
	unsigned int *devPrevDiag;
	unsigned int *devPPrevDiag;

	unsigned int *currDiag = new unsigned int[lenA + 1];
	unsigned int *prevDiag = new unsigned int[lenA + 1];
	unsigned int *pprevDiag = new unsigned int[lenA + 1];
	// Init first two diagonals
	pprevDiag[0] = 0;
	prevDiag[0] = 1;
	prevDiag[1] = 1;

	// CUDA Alloc
	hipMalloc((void **)&devA, (lenA + 1) * sizeof(char));
	hipMalloc((void **)&devB, (lenB + 1) * sizeof(char));
	hipMalloc((void **)&devCurrDiag, (lenA + 1) * sizeof(unsigned int));
	hipMalloc((void **)&devPrevDiag, (lenA + 1) * sizeof(unsigned int));
	hipMalloc((void **)&devPPrevDiag, (lenA + 1) * sizeof(unsigned int));

	// CUDA copy into device
	hipMemcpy((void *)devA, (void *)A, (lenA + 1) * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy((void *)devB, (void *)B, (lenB + 1) * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy((void *)devPPrevDiag, (void *)pprevDiag, (lenA + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy((void *)devPrevDiag, (void *)prevDiag, (lenA + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);

	for (int i = 3; i < 2 * (lenA + 1); i++)
	{
		int blockSize = min(i, 1024);
		int gridSize = ceil(float(i) / blockSize);

		editDistKernel<<<gridSize, blockSize>>>(devA, devB, lenA, lenB, devPPrevDiag, devPrevDiag, devCurrDiag, i);

		unsigned int *tmp = devPPrevDiag;
		devPPrevDiag = devPrevDiag;
		devPrevDiag = devCurrDiag;
		devCurrDiag = tmp;
	}
	// CUDA get result from device
	hipMemcpy((void *)&distance, (void *)&devPrevDiag[0], 1 * sizeof(unsigned int), hipMemcpyDeviceToHost);

	// CUDA free device memory
	hipFree(devA);
	hipFree(devB);
	hipFree(devPPrevDiag);
	hipFree(devPrevDiag);
	hipFree(devCurrDiag);

	return distance;
}

int main()
{
	int n = 10000;
	string A = generateWord(n);
	string B = generateWord(n);

	cout << "--------- STRING LENGTH = " << n << " ---------" << endl;
	// SEQUENTIAL
	auto start = system_clock::now();
	int distance = sequentialDistance(A, B);
	auto end = system_clock::now();
	auto elapsed = duration_cast<milliseconds>(end - start);
	cout << "Sequential [d=" << distance << "]: " << elapsed.count() << "ms" << endl;

	// PARALLEL
	start = system_clock::now();
	distance = parallelDistance(A.c_str(), B.c_str(), n, n);
	end = system_clock::now();
	elapsed = duration_cast<milliseconds>(end - start);
	cout << "Parallel [d=" << distance << "]: " << elapsed.count() << "ms" << endl;
}